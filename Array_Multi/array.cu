
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16;
const int blocksize = 16;

__global__
void multi(int *a, int *b)
{
	a[threadIdx.x] *= b[threadIdx.x];
}

int main()
{
    int a[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    int b[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};

    int *ad;
    int *bd;

    const int isize = N*sizeof(int);

    for(int i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n");

    hipMalloc( (void**)&ad, isize );
	hipMalloc( (void**)&bd, isize );

    hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    multi<<<dimGrid, dimBlock>>>(ad, bd);

    hipMemcpy( a, ad, isize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );

    for(int i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n");
}
