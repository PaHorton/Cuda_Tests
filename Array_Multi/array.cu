
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

const long N = pow(2,30);
const int blocksize = 16;

__global__
void GPU_multi(long *a, long *b) {
	a[threadIdx.x] *= b[threadIdx.x];
}

void CPU_multi(long *a, long *b) {
    for(long i = 0; i<N; i++){
        a[i] *= b[i];
    }
}


int main() {
    const long isize = N*sizeof(long);
    long *a = (long*) malloc(N*sizeof(long));
    long *b = (long*) malloc(N*sizeof(long));
    long *ad;
    long *bd;

    double cdiff = 0.0;
    double gdiff = 0.0;
    double gmdiff = 0.0;
    clock_t mstart, mstop, start, stop;

    for(long i = 0; i < N; i++){
        a[i] = i;
        b[i] = i;
    //    printf("%d ",a[i]);
    }
    //printf("\n");
    mstart = clock();
    hipMalloc( (void**)&ad, isize );
    hipMalloc( (void**)&bd, isize );

    hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    start = clock();
    GPU_multi<<<dimGrid, dimBlock>>>(ad, bd);
    stop = clock();
    hipMemcpy( a, ad, isize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );
    mstop = clock();
   // for(int i = 0; i < N; i++){
   //     printf("%d ",a[i]);
   // }
   // printf("\n");

    gdiff = (double) (stop - start)/CLOCKS_PER_SEC;
    gmdiff = (double) (mstop - mstart)/CLOCKS_PER_SEC ;
    for(long i = 0; i < N; i++){
        a[i] = i;
        b[i] = i;
    //    printf("%d ",a[i]);
    }
    // printf("\n");
    start = clock();
    CPU_multi(a,b);
    //for(int i = 0; i < N; i++){
    //    printf("%d ",a[i]);
    //}
    //printf("\n");
    stop = clock();
    cdiff = (double) (stop - start)/CLOCKS_PER_SEC;

    printf("Completed GPU multiplication of %ld in %.8f seconds\n", N, gdiff);
    printf("Completed CPU multiplication of %ld in %.8f seconds\n", N, cdiff);
    printf("GPU Memory moving time done in %.8f seconds\n", gmdiff);
    free(a);
    free(b);
    return EXIT_SUCCESS;
}
