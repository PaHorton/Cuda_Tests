#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

const int N = 16;
const int blocksize = 16;

__global__
void GPU_multi(int *a, int *b) {
	a[threadIdx.x] *= b[threadIdx.x];
}

void CPU_multi(int *a, int*b) {
    for(i = 0; i<N; i++){
        a[i] *= b[i]
    }
}


int main() {
    int a[N];
    int b[N];

    int *ad;
    int *bd;

    time_t start;
    time_t stop;

    const int isize = N*sizeof(int);

    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i;
        printf("%d ",a[i]);
    }
    printf("\n")
    time(&start);
    hipMalloc( (void**)&ad, isize );
	hipMalloc( (void**)&bd, isize );

    hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    GPU_multi<<<dimGrid, dimBlock>>>(ad, bd);

    hipMemcpy( a, ad, isize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );

    for(int i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n")
    time(&stop);
    diff = timediff(stop,start);
    printf("Completed GPU multiplication of %d in %d seconds\n", N, diff);

    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i;
        printf("%d ",a[i]);
    }
    printf("\n")
    time(&start);
    CPU_multi(a,b);
    for(int i = 0; i < N; i++){
        printf("%d ",a[i]);
    }
    printf("\n");
    time(&stop);
    diff = timediff(stop,start);
    printf("Completed CPU multiplication of %d in %d seconds\n", N, diff);
    return EXIT_SUCCESS;
}
